#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "knnquerycluster_cuda_kernel.h"

// Note: n is the clusters, m is the points
// Note: m >> n
// Note: nsample <= 200

// input: xyz (b, n, 3) xyz_idx (b, n) new_xyz (b, m, 3)
// output: idx (b, m, nsample) dist2 (b, m, nsample)
__global__ void knnquerycluster_cuda_kernel(int b, int n, int m, int nsample, const float *__restrict__ xyz, const int *__restrict__ xyz_idx, const float *__restrict__ new_xyz, int *__restrict__ idx, int *__restrict__ idx_abs, float *__restrict__ dist2) {
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= m) return;

    new_xyz += bs_idx * m * 3 + pt_idx * 3;
    xyz += bs_idx * n * 3;
    xyz_idx += bs_idx * n * 1;  // add xyz_idx initial position
    idx += bs_idx * m * nsample + pt_idx * nsample;
    idx_abs += bs_idx * m * nsample + pt_idx * nsample;

    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    //double* best = new double[nsample];
    //int* besti = new int[nsample];
    double best[200];
    int besti[200];
    for(int i = 0; i < nsample; i++) {
        best[i] = 1e40;
        besti[i] = 0;
    }
    for(int k = 0; k < n; k++){
        float x = xyz[k * 3 + 0];
        float y = xyz[k * 3 + 1];
        float z = xyz[k * 3 + 2];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
        for(int j = 0; j < nsample; j++){
            if(d2 < best[j]){
                for(int i = nsample - 1; i > j; i--){
                    best[i] = best[i - 1];
                    besti[i] = besti[i - 1];
                }
                best[j] = d2;
                besti[j] = k;
                break;
            }
        }
    }
    for(int i = 0; i < nsample; i++){
        //idx[i] = besti[i];
        idx[i] = xyz_idx[besti[i]];
        idx_abs[i] = besti[i];
        dist2[i] = best[i];
    }
    //delete []best;
    //delete []besti;
}


void knnquerycluster_cuda_launcher(int b, int n, int m, int nsample, const float *xyz, const int *xyz_idx, const float *new_xyz, int *idx, int *idx_abs, float *dist2, hipStream_t stream) {  // add xyz_idx
    // param new_xyz: (B, m, 3)
    // param xyz: (B, n, 3)
    // param xyz_idx: (B, n)     // add
    // param idx: (B, m, nsample)
    // param idx_abs: (B, m, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    knnquerycluster_cuda_kernel<<<blocks, threads, 0, stream>>>(b, n, m, nsample, xyz, xyz_idx, new_xyz, idx, idx_abs, dist2);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
